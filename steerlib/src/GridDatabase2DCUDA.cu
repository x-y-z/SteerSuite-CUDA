#include "hip/hip_runtime.h"
#include "griddatabase/GridDatabase2DCUDA.h"

#define BLOCKSIZE 16

#define MAX_FORCE_MAGNITUDE 3.0f
#define MAX_SPEED 1.3f
#define AGENT_MASS 1.0f

/*
Util::Vector vectorToGoal = _goalQueue.front().targetLocation - _position;

	// it is up to the agent to decide what it means to have "accomplished" or "completed" a goal.
	// for the simple AI, if the agent's distance to its goal is less than its radius, then the agent has reached the goal.
	if (vectorToGoal.lengthSquared() < _radius * _radius) {
		_goalQueue.pop();
		if (_goalQueue.size() != 0) {
			// in this case, there are still more goals, so start steering to the next goal.
			vectorToGoal = _goalQueue.front().targetLocation - _position;
		}
		else {
			// in this case, there are no more goals, so disable the agent and remove it from the spatial database.
			Util::AxisAlignedBox bounds(_position.x-_radius, _position.x+_radius, 0.0f, 0.0f, _position.z-_radius, _position.z+_radius);
			gSpatialDatabase->removeObject( this, bounds);
			_enabled = false;
			return;
		}
	}

	// use the vectorToGoal as a force for the agent to steer towards its goal.
	// the euler integration step will clamp this vector to a reasonable value, if needed.
	// also, the Euler step updates the agent's position in the spatial database.
	_doEulerStep(vectorToGoal, dt);


	// compute acceleration, _velocity, and newPosition by a simple Euler step
	const Util::Vector clippedForce = Util::clamp(steeringDecisionForce, MAX_FORCE_MAGNITUDE);
	Util::Vector acceleration = (clippedForce / AGENT_MASS);
	_velocity = _velocity + (dt*acceleration);
	_velocity = clamp(_velocity, MAX_SPEED);  // clamp _velocity to the max speed
	const Util::Point newPosition = _position + (dt*_velocity);

	// For this simple agent, we just make the orientation point along the agent's current velocity.
	if (_velocity.lengthSquared() != 0.0f) {
		_forward = normalize(_velocity);
	}

	// update the database with the new agent's setup
	Util::AxisAlignedBox oldBounds(_position.x - _radius, _position.x + _radius, 0.0f, 0.0f, _position.z - _radius, _position.z + _radius);
	Util::AxisAlignedBox newBounds(newPosition.x - _radius, newPosition.x + _radius, 0.0f, 0.0f, newPosition.z - _radius, newPosition.z + _radius);
	gSpatialDatabase->updateObject( this, oldBounds, newBounds);

	_position = newPosition;
	*/

__global__ void updateAI_kernel(cuda_item *cudaItems, float currentSimulationTime, float dt, unsigned int currentFrameNumbers,
								int agentNum, int obstacleNum, int *disabledAgents)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	

	if (x >= agentNum)
		return;

	if (!cudaItems[x]._agent._enabled)
	{
		cudaItems[x].type = -1;
		disabledAgents[x] = 1;
		return;
	}
	//printf("thread: %d\n", x);

	int curGoal = cudaItems[x]._agent._curGoal;
	float radius = cudaItems[x]._agent._radius;
	float3 position = cudaItems[x]._agent._position;
	float3 vectorToGoal = cudaItems[x]._agent._goalQueue[curGoal] - cudaItems[x]._agent._position;

	//printf("goal: (%f,%f,%f)\n", cudaItems[x]._agent._goalQueue[curGoal].x, cudaItems[x]._agent._goalQueue[curGoal].y, cudaItems[x]._agent._goalQueue[curGoal].z);
	//printf("position: (%f,%f,%f)\n", cudaItems[x]._agent._position.x, cudaItems[x]._agent._position.y, cudaItems[x]._agent._position.z);

	//printf("vector to Goal: (%f,%f,%f), radius: %f\n", vectorToGoal.x, vectorToGoal.y, vectorToGoal.z, radius);

	cudaItems[x]._agent._usedGoal = 0;

	// it is up to the agent to decide what it means to have "accomplished" or "completed" a goal.
	// for the simple AI, if the agent's distance to its goal is less than its radius, then the agent has reached the goal.
	if (dot(vectorToGoal, vectorToGoal) < radius * radius) {
		cudaItems[x]._agent._curGoal++;
		cudaItems[x]._agent._usedGoal++;
		if (cudaItems[x]._agent._curGoal != cudaItems[x]._agent._goalQSize) {
			// in this case, there are still more goals, so start steering to the next goal.
			vectorToGoal = cudaItems[x]._agent._goalQueue[cudaItems[x]._agent._curGoal] - cudaItems[x]._agent._position;
		}
		else {
			// in this case, there are no more goals, so disable the agent and remove it from the spatial database.
			AABox bounds = {position.x-radius, position.x+radius, 0.0f, 0.0f, position.z-radius, position.z+radius};
			//gSpatialDatabase->removeObject( this, bounds);
			cudaItems[x]._agent._newBounds = bounds;
			cudaItems[x]._agent._enabled = false;
			disabledAgents[x] = 1;
			//printf("disabled one\n");
			return;
		}
	}
	
	float3 clippedForce = clamp(vectorToGoal, MAX_FORCE_MAGNITUDE);
	float3 acceleration = (clippedForce / AGENT_MASS);
	cudaItems[x]._agent._velocity += (dt*acceleration);
	cudaItems[x]._agent._velocity = clamp(cudaItems[x]._agent._velocity, MAX_SPEED);  // clamp _velocity to the max speed
	float3 newPosition = cudaItems[x]._agent._position + (dt*cudaItems[x]._agent._velocity);

	// For this simple agent, we just make the orientation point along the agent's current velocity.
	if (dot(cudaItems[x]._agent._velocity,cudaItems[x]._agent._velocity) != 0.0f) {
		cudaItems[x]._agent._forward = normalize(cudaItems[x]._agent._velocity);
	}

	// update the database with the new agent's setup
	AABox oldBounds = {cudaItems[x]._agent._position.x - cudaItems[x]._agent._radius, 
					   cudaItems[x]._agent._position.x + cudaItems[x]._agent._radius, 
					   0.0f, 0.0f, 
					   cudaItems[x]._agent._position.z - cudaItems[x]._agent._radius, 
					   cudaItems[x]._agent._position.z + cudaItems[x]._agent._radius};
	AABox newBounds = {newPosition.x - cudaItems[x]._agent._radius, 
		               newPosition.x + cudaItems[x]._agent._radius, 
					   0.0f, 0.0f, 
					   newPosition.z - cudaItems[x]._agent._radius, 
					   newPosition.z + cudaItems[x]._agent._radius};
	//gSpatialDatabase->updateObject( this, oldBounds, newBounds);

	cudaItems[x]._agent._oldBounds = oldBounds;
	cudaItems[x]._agent._newBounds = newBounds;

	cudaItems[x]._agent._position = newPosition;

}

void launch_updateAICUDA(cuda_item *cudaItems, float currentSimulationTime, float simulatonDt, unsigned int currentFrameNumber, 
	                     int agentNum, int obstacleNum, int &numDisabledAgents)
{
	dim3 block(BLOCKSIZE*BLOCKSIZE);
	dim3 grid((agentNum)/(BLOCKSIZE*BLOCKSIZE) + 1);

	int *disAgents, *hostDisAgents;
	CudaSafeCall(hipMalloc(&disAgents, sizeof(int)*agentNum));
	CudaSafeCall(hipMemset(disAgents,0, sizeof(int)*agentNum));

	hostDisAgents = new int[agentNum];

	updateAI_kernel<<<grid, block>>>(cudaItems, currentSimulationTime, simulatonDt, currentFrameNumber,
		                             agentNum, obstacleNum, disAgents);

	hipError_t res = hipDeviceSynchronize();

	res = (hipMemcpy(hostDisAgents, disAgents, sizeof(int)*agentNum, hipMemcpyDeviceToHost));

	for (int i = 0; i < agentNum; ++i)
	{
		numDisabledAgents += hostDisAgents[i];
	}


}